
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void initWith(float num, float *a, int N)
{
  int i = threadIdx.x + blockIdx.x * blockDim.x;
  if(i < N)
    a[i] = num;
}

__global__ void addVectorsInto(float *result, float *a, float *b, int N)
{
  int i = threadIdx.x + blockIdx.x * blockDim.x;
  if(i < N)
    result[i] = a[i] + b[i];
}

void checkElementsAre(float target, float *array, int N)
{
  for(int i = 0; i < N; i++)
  {
    if(array[i] != target)
    {
      printf("FAIL: array[%d] - %0.0f does not equal %0.0f\n", i, array[i], target);
      exit(1);
    }
  }
  printf("SUCCESS! All values added correctly.\n");
}

int main()
{
  const int N = 2<<20;
  size_t size = N * sizeof(float);

  float *a;
  float *b;
  float *c;

  hipMallocManaged(&a, size);
  hipMallocManaged(&b, size);
  hipMallocManaged(&c, size);
  
  size_t thread_per_block = 1024;
  size_t number_of_blocks = (N + thread_per_block - 1) / thread_per_block;
  
  initWith<<<number_of_blocks, thread_per_block>>>(3, a, N);
  initWith<<<number_of_blocks, thread_per_block>>>(4, b, N);
  initWith<<<number_of_blocks, thread_per_block>>>(0, c, N);

  addVectorsInto<<<number_of_blocks, thread_per_block>>>(c, a, b, N);
  hipDeviceSynchronize();
  checkElementsAre(7, c, N);

  hipFree(a);
  hipFree(b);
  hipFree(c);
}
